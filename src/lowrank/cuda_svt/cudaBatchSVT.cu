#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <assert.h>
#include <string.h>
#include "cudacomplex.h"
#include "cudaBatchSVT.h"
#include "deviceSVD.cuh" 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include "num/cuda_commons.h"
#include <iostream>

#ifndef BATCH_SVT_USE_FIXED_SIZE
#include "nvrtc_batchCHQL.cuh"
#endif

// events for timing
static hipEvent_t startEvent, stopEvent;
static float ms;
static hipblasStatus_t ret;
extern hipblasHandle_t cublas_handle;

void init_batch_svt(int devId)
{
  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );
}

__attribute__((constructor)) static void init() 
{
  init_batch_svt(0); 
}

void cublas_transpose(hipblasHandle_t handle, int m, int n, const hipComplex* d_a, hipComplex* d_at)
{
  //Transpose
  const hipComplex alpha = make_hipComplex(1.0f,0.0f);
  const hipComplex beta  = make_hipComplex(0.0f,0.0f);
  //Perform operation with cublas
  checkCuda( hipEventRecord(startEvent, 0) );
  ret = hipblasCgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                    m, n, 
                    &alpha, d_a, n, &beta, NULL, m,
                    d_at, m);

  if (ret != HIPBLAS_STATUS_SUCCESS)
  {
      printf("cublasCgeamm returned error code %d, line(%d)\n", ret, __LINE__);
      exit(EXIT_FAILURE);
  }
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Transpose took %f\n", ms);
}

void cublas_transpose_float(hipblasHandle_t handle, int m, int n, const float* d_a, float* d_at)
{
  //Transpose
  const float alpha = 1.0f;
  const float beta  = 0.0f;
  //Perform operation with cublas
  checkCuda( hipEventRecord(startEvent, 0) );
  ret = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                    m, n, 
                    &alpha, d_a, n, &beta, NULL, m,
                    d_at, m);

  if (ret != HIPBLAS_STATUS_SUCCESS)
  {
      printf("cublasSgeamm returned error code %d, line(%d)\n", ret, __LINE__);
      exit(EXIT_FAILURE);
  }
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("Transpose float took %f\n", ms);
}

void cublas_batch_dgemm(hipblasHandle_t handle, 
                        hipblasOperation_t transa, hipblasOperation_t transb, 
                        int m, int n, int k, 
                        const hipComplex** a, int lda, 
                        const hipComplex** b, int ldb, 
                        hipComplex **c, int ldc, 
                        int batch_size)
{
  //Transpose
  const hipComplex alpha = make_hipComplex(1.0f,0.0f);
  const hipComplex beta  = make_hipComplex(0.0f,0.0f);
  //Perform operation with cublas
  checkCuda( hipEventRecord(startEvent, 0) );
  ret = hipblasCgemmBatched(handle, transa, transb, 
                    m, n, k,
                    &alpha, 
                    a, lda, 
                    b, ldb,
                    &beta, 
                    c, ldc,
                    batch_size);

  if (ret != HIPBLAS_STATUS_SUCCESS)
  {
      printf("hipblasCgemm returned error code %d, line(%d)\n", ret, __LINE__);
      exit(EXIT_FAILURE);
  }
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );
  checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
  printf("batch multiplication took %f\n", ms);
}

#define CREATE_BATCH_PTR(start, step, batch_size, dst) create_batch_ptr((void*)start, step, batch_size, (void**)dst)
void create_batch_ptr(void* start, size_t step, int batch_size, void** dst)
{
  int batch_ptrs_size = sizeof(void*) * batch_size;
  void** ptrs = (void**)malloc(batch_ptrs_size);
  for(int i=0; i < batch_size; i++)
  {
    ptrs[i] = ((char*)start) + step * i;
  }
  checkCuda( hipMemcpy(dst, ptrs, batch_ptrs_size, hipMemcpyHostToDevice) );
}

/*TODO include debug routine somwhere*/
// void WriteMatrixDevice(singlecomplex *matrix, int m, int n)
// {
//   int size = sizeof(singlecomplex) * m * n;
//   singlecomplex *host_copy = (singlecomplex*)malloc(size);
//   checkCuda( hipMemcpy(host_copy, matrix, size, hipMemcpyDeviceToHost) );
//   hipDeviceSynchronize();
//   WriteMatrix ((singlecomplex(*)[maxA])host_copy, m, n);
//   free(host_copy);
// }

// void WriteDiagDevice(float *diag, int m)
// {
//   int size = sizeof(float) * m;
//   float *host_copy = (float*)malloc(size);
//   checkCuda( hipMemcpy(host_copy, diag, size, hipMemcpyDeviceToHost) );
//   for(int i=0; i < m; i++) {
//     printf("%f\n", host_copy[i]);
//   }
//   free(host_copy);
// }

void cuda_batch_svt(int m, int n, int batch_size, float tau, _Complex float *_d_tau_a, const _Complex float *_d_a)
{
  const singlecomplex *d_a = (const singlecomplex*)_d_a;
  singlecomplex *d_tau_a = (singlecomplex*)_d_tau_a;

#ifdef BATCH_SVT_USE_FIXED_SIZE
  assert(m == maxA && "Batch SVT is compiled with fixed size, and you passed in something different");
  assert(batch_size == BATCH_SIZE && "Batch SVT is compiled with fixed size, and you passed in something different");
#else

#endif
  bool m_gt_n = m > n;
  if(m_gt_n) {
    int tmp = n;
    n = m;
    m = tmp;
  }
  const long block_a_mem_size = m * n * sizeof(singlecomplex);
  const long block_b_mem_size = m * m * sizeof(singlecomplex);
  const long block_s_mem_size = m * sizeof(singlecomplex);
  const long b_mem_size = m * m * sizeof(singlecomplex) * batch_size;
  const long s_mem_size = m * sizeof(singlecomplex) * batch_size;
  const long batch_ptrs_size = sizeof(const hipComplex*) * batch_size;

  /* Compute B = A * A^T */
  singlecomplex *d_b;
  const hipComplex **d_a_ptr; 
  hipComplex **d_b_ptr;
  checkCuda( hipMalloc(&d_b, b_mem_size) );
  checkCuda( hipMalloc(&d_a_ptr, batch_ptrs_size) );
  checkCuda( hipMalloc(&d_b_ptr, batch_ptrs_size) );
  CREATE_BATCH_PTR(d_a, block_a_mem_size, batch_size, d_a_ptr);
  CREATE_BATCH_PTR(d_b, block_b_mem_size, batch_size, d_b_ptr);
  if(m_gt_n) {
    cublas_batch_dgemm(cublas_handle, HIPBLAS_OP_C, HIPBLAS_OP_N, m, m, n, d_a_ptr, n, d_a_ptr, n, d_b_ptr, m, batch_size);
  } else {
    cublas_batch_dgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_C, m, m, n, d_a_ptr, m, d_a_ptr, m, d_b_ptr, m, batch_size);
  }
  checkCuda( hipFree(d_b_ptr) );

  /* Enter interleaved mode */
  singlecomplex *d_bt;
  checkCuda( hipMalloc(&d_bt, b_mem_size) );
  cublas_transpose(cublas_handle, batch_size, m * m, (const hipComplex*)d_b, (hipComplex*)d_bt);

  /* Compute Core SVD */
  singlecomplex *d_ut = d_b;
  float *d_st, *d_work;
  checkCuda( hipMalloc(&d_st, s_mem_size) );
  checkCuda( hipMalloc(&d_work, s_mem_size * 3) );
#ifdef BATCH_SVT_USE_FIXED_SIZE
  batchCHQL<<<batch_size/INTERLEAVE,INTERLEAVE>>>(d_bt, m, d_st, d_ut, d_work);
#else
  nvrtc_batchCHQL(d_bt, m, d_st, d_ut, d_work, batch_size);
#endif 

  /* Exit interleaved mode */
  singlecomplex *d_u = d_bt;
  float *d_s;
  checkCuda( hipMalloc(&d_s, s_mem_size) );
  cublas_transpose(cublas_handle, m * m, batch_size, (const hipComplex*)d_ut, (hipComplex*)d_u);
  cublas_transpose_float(cublas_handle, m, batch_size, (const float*)d_st, (float*)d_s);

  /* Compute U * tau(S) * U^T. Currently this is not optimal */
  singlecomplex *d_u_scal_s;
  singlecomplex *d_usut = d_ut;
  checkCuda( hipMalloc(&d_u_scal_s, b_mem_size) );
  checkCuda( hipMemcpy(d_u_scal_s, d_u, b_mem_size, hipMemcpyDeviceToDevice) );
  batch_clascl_diag_thres_full(m, m, batch_size, d_s, d_u_scal_s, m, tau);
  checkCuda( hipFree(d_s) );

  const hipComplex **d_u_scal_s_ptr,  **d_u_ptr;
  hipComplex **dusut_ptr;
  checkCuda( hipMalloc(&d_u_scal_s_ptr, batch_ptrs_size) );
  checkCuda( hipMalloc(&d_u_ptr, batch_ptrs_size) );
  checkCuda( hipMalloc(&dusut_ptr, batch_ptrs_size) );
  CREATE_BATCH_PTR(d_u_scal_s, block_b_mem_size, batch_size, d_u_scal_s_ptr);
  CREATE_BATCH_PTR(d_u, block_b_mem_size, batch_size, d_u_ptr);
  CREATE_BATCH_PTR(d_usut, block_b_mem_size, batch_size, dusut_ptr);
  cublas_batch_dgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_C, m, m, m, d_u_scal_s_ptr, m, d_u_ptr, m, dusut_ptr, m, batch_size);

  /* Compute tau(USU^T) * A */
  hipComplex **d_tau_a_ptr;
  checkCuda( hipMalloc(&d_tau_a_ptr, batch_ptrs_size) );
  CREATE_BATCH_PTR(d_tau_a, block_a_mem_size, batch_size, d_tau_a_ptr);
  if(m_gt_n) {
    cublas_batch_dgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T, 
                      n, m, m, 
                      (const hipComplex**)d_a_ptr, n, 
                      (const hipComplex**)dusut_ptr, m, 
                      d_tau_a_ptr, n, batch_size);
  } else {
    cublas_batch_dgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                      m, n, m, 
                      (const hipComplex**)dusut_ptr, m, 
                      (const hipComplex**)d_a_ptr, m, 
                      d_tau_a_ptr, m, batch_size);
  }

  checkCuda( hipFree(d_u_ptr) );
  checkCuda( hipFree(d_usut) );
  checkCuda( hipFree(d_u_scal_s_ptr) );
  checkCuda( hipFree(dusut_ptr));
  checkCuda( hipFree(d_tau_a_ptr));
  checkCuda( hipFree(d_a_ptr) );
  checkCuda( hipFree(d_u) );
}
